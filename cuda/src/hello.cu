#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "matrix.h"

#define THD_PER_BLK 256
#define CEIL_DIV(A, B) ((A+B-1) / B) 



static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void helloCUDA()
{
    printf("Hello CUDA %d\n", threadIdx.x + blockIdx.x * blockDim.x);
}

__global__ void init(mat_t * A, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N )
    {
        // hiprandState state;
        // hiprand_init(clock64(), i, 0, &state);
        // A[i] = hiprand_uniform(&state) * 100;
        A[i] = 1.0f / (i+1)  * 100;
    }
}

__global__ void mmult_stride(mat_t * A, mat_t * B, mat_t * C, int m, int n, int p)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int total_threads = blockDim.x * gridDim.x;

    int num_elems = m * p;


    for (; i < num_elems; i += total_threads)
    {
        int rowA = i / m;
        int colB = i % p;
        int sum = 0;
        for (int j = 0; j < n; j++)
        {
            sum += A[j + rowA * n]  * B[j*p + colB];
        }
        C[i] = sum;
    }

}

__global__ void mmult_nostride(mat_t * A, mat_t * B, mat_t * C, int m, int n, int p)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // int total_threads = blockDim.x * gridDim.x;

    int num_elems = m * p;
    int rowA = i / m;
    int colB = i % p;
    int sum = 0;
    if (i < num_elems)
    {
        for (int j = 0; j < n; j++)
        {
            sum += A[j + rowA * n]  * B[j*p + colB];
        }
        C[i] = sum;
    }

}

__global__ void sum(mat_t * A, mat_t * B, mat_t * C, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N )
    {
        C[i] = A[i] + B[i];
    }
    
}

int main()
{
    hipDeviceProp_t prop;   
    hipGetDeviceProperties( &prop, 0);
    printf("Device: %s\n%d threads per block\n%d per MP\n", prop.name, prop.maxThreadsPerBlock, prop.maxThreadsPerMultiProcessor);
    printf("%d total multiprocessors\n\n", prop.multiProcessorCount);
    int block_count = prop.multiProcessorCount * prop.maxThreadsPerMultiProcessor / prop.maxThreadsPerBlock;
    printf("Using %d blocks\n", block_count);
    
    unsigned int n = 1000;
    unsigned int m = 1000;
    unsigned int p = 1000;
    
    mat_t * A, * B, * C;
    mat_t * h_A, * h_B, * h_C;

    h_A = (mat_t *) malloc(m * n * sizeof(mat_t));
    h_B = (mat_t *) malloc(n * p * sizeof(mat_t));
    h_C = (mat_t *) malloc(m * p * sizeof(mat_t));

    hipMalloc ((void**)&A, m * n * sizeof(mat_t));
    hipMalloc ((void**)&B, n * p * sizeof(mat_t));
    hipMalloc ((void**)&C, m * p * sizeof(mat_t));

    float elapsed=0;
    hipEvent_t start, stop;

    init    <<<CEIL_DIV(m * n, THD_PER_BLK), THD_PER_BLK>>> (A,m * n);
    init    <<<CEIL_DIV(n * p, THD_PER_BLK), THD_PER_BLK>>> (B,n * p);
    // sum     <<<CEIL_DIV(n, THD_PER_BLK), THD_PER_BLK>>> (A,B,C,n);

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR( hipEventRecord(start, 0));
    mmult_nostride   <<<CEIL_DIV(m * p, THD_PER_BLK), THD_PER_BLK>>> (A,B,C, n, m, p);
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize (stop) );
    HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop) );
    printf("GPU NoStride: %f ms\n", elapsed);
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR( hipEventRecord(start, 0));
    mmult_stride   <<<block_count,prop.maxThreadsPerBlock>>> (A,B,C, n, m, p);
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize (stop) );
    HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop) );
    printf("GPU Stride: %f ms\n", elapsed);

    hipMemcpy(h_A, A, n * sizeof(A), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, B, n * sizeof(B), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, C, n * sizeof(C), hipMemcpyDeviceToHost);

    hipError_t err = hipDeviceSynchronize();

    hipFree(A);
    hipFree(B);
    hipFree(C);
    // printf("%f | %f \n%f | %f\n\n", h_A[0], h_A[1], h_A[2], h_A[3]);
    // printf("%f | %f \n%f | %f\n\n", h_B[0], h_B[1], h_B[2], h_B[3]);
    // printf("%f | %f \n%f | %f\n\n", h_C[0], h_C[1], h_C[2], h_C[3]);

    free(h_A);
    free(h_B);
    free(h_C);

    

    return 0;
}