#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "matrix.h"

#define THD_PER_BLK 256
#define L1_$LINE_BYTES 128
#define TILESIZE (L1_$LINE_BYTES / sizeof(mat_t) )
#define CEIL_DIV(A, B) ((A+B-1) / B) 

hipEvent_t start, stop;
float elapsed=0;

#define TIME(f, msg)                            \
hipEventCreate(&start);                        \
hipEventCreate(&stop);                         \
hipEventRecord(start, 0);                      \
f;                                              \
hipEventRecord(stop, 0);                       \
hipEventSynchronize (stop);                    \
hipEventElapsedTime(&elapsed, start, stop);    \
printf(msg": %f ms\n", elapsed);    

__global__ void helloCUDA()
{
    printf("Hello CUDA %d\n", threadIdx.x + blockIdx.x * blockDim.x);
}

__global__ void init(mat_t * A, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N )
    {
        // hiprandState state;
        // hiprand_init(clock64(), i, 0, &state);
        // A[i] = hiprand_uniform(&state) * 100;
        A[i] = 1.0f / (i+1)  * 100;
    }
}

__global__ void mmult_stride(mat_t * A, mat_t * B, mat_t * C, int m, int n, int p)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int total_threads = blockDim.x * gridDim.x;

    int num_elems = m * p;


    for (; i < num_elems; i += total_threads)
    {
        int rowA = i / m;
        int colB = i % p;
        int sum = 0;
        for (int j = 0; j < n; j++)
        {
            sum += A[j + rowA * n]  * B[j*p + colB];
        }
        C[i] = sum;
    }

}

__global__ void mmult_nostride(mat_t * A, mat_t * B, mat_t * C, int m, int n, int p)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // int total_threads = blockDim.x * gridDim.x;

    int num_elems = m * p;
    int rowA = i / m;
    int colB = i % p;
    int sum = 0;
    if (i < num_elems)
    {
        for (int j = 0; j < n; j++)
        {
            sum += A[j + rowA * n]  * B[j*p + colB];
        }
        C[i] = sum;
    }

}

// // kernel call to tiling implementation. TILESIZE is likely 32 on a side. 
// __global__ void mmult_tiling(mat_t * A, mat_t * B, mat_t * C, int m, int n, int p)
// {
//     __shared__ mat_t s_A[TILESIZE][TILESIZE];
//     __shared__ mat_t s_B[TILESIZE][TILESIZE];
    
    
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
//     int idy = threadIdx.y + blockIdx.y * blockDim.y;

//     int tilex = blockIdx.x;
//     int tiley = blockIdx.y;
    
//     int offx = threadIdx.x + tilex * blockDim.x; // row of Mat A
//     int offy = threadIdx.y + tiley * blockDim.y; // row of Mat B
    
//     for(int i = 0; i < TILESIZE; i++)
//     {
//         s_A[tiley][tilex] = A[row*width + (i*2 + tx)];
//         s_B[tiley][tilex] = B[(i*2 + ty)*width + col];
//         __syncthreads();

//     }    
// }

__global__ void sum(mat_t * A, mat_t * B, mat_t * C, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if ( i < N )
    {
        C[i] = A[i] + B[i];
    }
    
}

int main()
{
    hipDeviceProp_t prop;   
    hipGetDeviceProperties( &prop, 0);
    printf("Device: %s\n%d threads per block\n%d per MP\n", prop.name, prop.maxThreadsPerBlock, prop.maxThreadsPerMultiProcessor);
    printf("%d total multiprocessors\n", prop.multiProcessorCount);
    int block_count = prop.multiProcessorCount * prop.maxThreadsPerMultiProcessor / prop.maxThreadsPerBlock;
    printf("Using %d blocks\n", block_count);
    // printf("l1 Cache size: %ld\n", prop.sharedMemPerBlock);
    printf("\n");
    
    unsigned int m = 2000;
    unsigned int n = 2000;
    unsigned int p = 2000;
    
    mat_t * A, * B, * C, * D;
    mat_t * h_A, * h_B, * h_C, * h_D;

    h_A = (mat_t *) malloc(m * n * sizeof(mat_t));
    h_B = (mat_t *) malloc(n * p * sizeof(mat_t));
    h_C = (mat_t *) malloc(m * p * sizeof(mat_t));
    h_D = (mat_t *) malloc(m * p * sizeof(mat_t));

    hipMalloc ((void**)&A, m * n * sizeof(mat_t));
    hipMalloc ((void**)&B, n * p * sizeof(mat_t));
    hipMalloc ((void**)&C, m * p * sizeof(mat_t));
    hipMalloc ((void**)&D, m * p * sizeof(mat_t));

    init    <<<CEIL_DIV(m * n, THD_PER_BLK), THD_PER_BLK>>> (A,m * n);
    init    <<<CEIL_DIV(n * p, THD_PER_BLK), THD_PER_BLK>>> (B,n * p);
    // sum     <<<CEIL_DIV(n, THD_PER_BLK), THD_PER_BLK>>> (A,B,C,n);

    TIME((mmult_nostride   <<<CEIL_DIV(m * p, 256), 256>>> (A,B,C, m, n, p)), "GPU NOSTRIDE 256");
    TIME((mmult_nostride   <<<CEIL_DIV(m * p, 512), 512>>> (A,B,C, m, n, p)), "GPU NOSTRIDE 512");
    TIME((mmult_nostride   <<<CEIL_DIV(m * p, 1024), 1024>>> (A,B,C, m, n, p)), "GPU NOSTRIDE 1024");

    TIME((mmult_stride   <<<block_count,256>>> (A,B,D, m, n, p)), "GPU STRIDE 32x256");
    TIME((mmult_stride   <<<block_count,512>>> (A,B,D, m, n, p)), "GPU STRIDE 32x512");
    TIME((mmult_stride   <<<block_count,1024>>> (A,B,D, m, n, p)), "GPU STRIDE 32x1024");

    // TIME((mmult_tiling   <<<(m / TILESIZE, p / TILESIZE),(TILESIZE, TILESIZE)>>> (A,B,C, m, n, p)), "GPU Tiled");

    hipMemcpy(h_A, A, m * n * sizeof(mat_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, B, n * p * sizeof(mat_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, C, m * p * sizeof(mat_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_D, D, m * p * sizeof(mat_t), hipMemcpyDeviceToHost);
    printf("\tError: %.3f\n", test(h_D, h_C, m, p));
    hipError_t err = hipDeviceSynchronize();

    hipFree(A);
    hipFree(B);
    hipFree(C);
    // printf("%f | %f \n%f | %f\n\n", h_A[0], h_A[1], h_A[2], h_A[3]);
    // printf("%f | %f \n%f | %f\n\n", h_B[0], h_B[1], h_B[2], h_B[3]);
    // printf("%f | %f \n%f | %f\n\n", h_C[0], h_C[1], h_C[2], h_C[3]);

    TIME(ompMatMult(h_A,h_B,h_D, m, n, p, 2), "OMP 2");
    printf("\tError: %.3f\n", test(h_D, h_C, m, p));
    TIME(ompMatMult(h_A,h_B,h_D, m, n, p, 4), "OMP 4");
    printf("\tError: %.3f\n", test(h_D, h_C, m, p));
    TIME(ompMatMult(h_A,h_B,h_D, m, n, p, 8), "OMP 8");
    printf("\tError: %.3f\n", test(h_D, h_C, m, p));
    
    TIME(pthread_matmult(h_A,h_B,h_D, m, n, p, 2), "PTHREAD 2");
    printf("\tError: %.3f\n", test(h_D, h_C, m, p));
    TIME(pthread_matmult(h_A,h_B,h_D, m, n, p, 4), "PTHREAD 4");
    printf("\tError: %.3f\n", test(h_D, h_C, m, p));
    TIME(pthread_matmult(h_A,h_B,h_D, m, n, p, 8), "PTHREAD 8");
    printf("\tError: %.3f\n", test(h_D, h_C, m, p));

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);

    return 0;
}